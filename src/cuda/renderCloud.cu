#include "hip/hip_runtime.h"
#include "cudaTypeDef.cuh"

__global__ void makeCloud(const cv::cuda::PtrStep<float> depthImg,const cv::cuda::PtrStep<uchar3> colorImg,const Eigen::Matrix3f intrinsic_inv,
                          const int rows, const int cols, float* ptrCloud){
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;
    if(x < cols && y < rows){
        const int indexStart = (y * cols + x) * 7;
        const float depth = depthImg.ptr(y)[x];
        if(depth <= 0){
            ptrCloud[indexStart] = 0.f;
            ptrCloud[indexStart + 1] = 0.f;
            ptrCloud[indexStart + 2] = 0.f;
            ptrCloud[indexStart + 3] = 0.f;
            ptrCloud[indexStart + 4] = 0.f;
            ptrCloud[indexStart + 5] = 0.f;
            ptrCloud[indexStart + 6] = 0.f;
            return;
        }
        Eigen::Vector3f vertex(x * depth, y * depth, depth);
        Eigen::Vector3f result = intrinsic_inv * vertex;
        ptrCloud[indexStart] = result(0,0);
        ptrCloud[indexStart + 1] = result(1,0);
        ptrCloud[indexStart + 2] = result(2,0);
        ptrCloud[indexStart + 3] = colorImg.ptr(y)[x].z;
        ptrCloud[indexStart + 4] = colorImg.ptr(y)[x].y;
        ptrCloud[indexStart + 5] = colorImg.ptr(y)[x].x;
        ptrCloud[indexStart + 6] = 1.0f;
    }
}

extern "C" void renderCloud_CUDA(const cv::cuda::GpuMat & depthImg, const cv::cuda::GpuMat & colorImg, const Eigen::Matrix3f & intrinsic_inv,
                            float* ptrCloud){
    dim3 block(32, 8);
    dim3 grid((depthImg.cols + block.x - 1) / block.x, (depthImg.rows + block.y - 1) / block.y);
    makeCloud<<<grid,block>>>(depthImg, colorImg, intrinsic_inv, depthImg.rows, depthImg.cols, ptrCloud);
    hipDeviceSynchronize();
}
